#include "hip/hip_runtime.h"

#include <optix_world.h>

#include "core/optix_global.h"
#include "core/Ray.h"
#include "core/Payload.h"
#include "samplers/Util.h"
#include "core/RNG.h"
#include "core/ONB.h"
#include "samplers/Random2D.h"
#include <optix_device.h>

rtBuffer<RNG> rngs;
//rtBuffer<float2> u;
rtBuffer<float3> coeff;
rtBuffer<unsigned int> ns;
rtDeclareVariable(int, N, ,);
rtDeclareVariable(uint, pixelIdx, rtLaunchIndex, );
rtDeclareVariable(float3, sensorNormal, , );
rtDeclareVariable(float3, sensorPos, , );
rtDeclareVariable(rtObject, root, , );
rtDeclareVariable(unsigned int, NskyPatches, , );

RT_PROGRAM void sensor(void) {

	int nSamples = 500;
	ReinhartPayload pl;
	pl.rng = rngs[pixelIdx];

	Random2D sampler(&pl.rng, nSamples);
	float2 unifSample; //= u[pixelIdx];

	while (sampler.Next2D(&unifSample)) {

		pl.value = make_float3(0.0f);
		pl.depth = 0;


		ONB onb(sensorNormal);
		float3 dir   = UniformHemisphereSample(unifSample.x, unifSample.y);
		float3 dirW  = onb.LocalToWorld(dir);
		Ray ray = make_Ray(sensorPos, dirW, RayType::RADIANCE, 0, RT_DEFAULT_MAX);
		//rtPrintf("%f %f %f\n", dirW.x, dirW.y, dirW.z);

		rtTrace(root, ray, pl);
		
		if (fmaxf(pl.value) > 0.0f) {
			float3 value = pl.value  ;
			//rtPrintf("%f %f\n", coeff[pl.patchID].x, value.x);
			atomicAdd(&coeff[pl.patchID].x, value.x);
			atomicAdd(&coeff[pl.patchID].y, value.y);
		    atomicAdd(&coeff[pl.patchID].z, value.z);
			//ns[pl.patchID]++;
		}
		
		
	}
	/*for (int i = 0; i < NskyPatches; i++) {
		coeff[i].x /= ns[i];
		coeff[i].y /= ns[i];
		coeff[i].z /= ns[i];
		coeff[i] = coeff[i] * M_PI;
		//rtPrintf("%d %f\n", ns[i], coeff[i].x);
	}*/
	rngs[pixelIdx] = pl.rng;

}


RT_PROGRAM void exception() {

	rtPrintExceptionDetails();

}
