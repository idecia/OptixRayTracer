#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "Camera.h"
#include "Film.h"


using namespace optix;

rtBuffer<float3, 2>   outputBuffer;
rtBuffer<RNG, 2> rngs;
rtDeclareVariable(uint2, idx, rtLaunchIndex, );
rtDeclareVariable(Camera, camera, , );
rtDeclareVariable(Film, film, , );
rtDeclareVariable(rtObject, root, , );

struct Payload {

	RNG rng;
	float3 color;

};

RT_PROGRAM void pinhole(void) {
	Payload payload;
	payload.rng = rngs[idx];
	Random2D sampler(&payload.rng, 1u);
	float2 sample;
	while (sampler.Next2D(&sample)) {
		float2 filmSample = film.Sample(idx, sample)
		Ray ray = camera.GenerateRay(filmSample);
		rtTrace(root, ray, payload);
		film.PutSample(filmSample, payload.color)
	}
	rngs[index] = payload.rng;

}