#include "hip/hip_runtime.h"
#include "lights/AllLights.h"
#include "core/Ray.h"
#include "samplers/Random2D.h"
#include "core/Payload.h"
#include "bxdfs/Lambertian.h"
#include "core/optix_global.h"
#include "core/ONB.h"
#include <optix_device.h>

#define MAX_DEPTH 1

rtDeclareVariable(uint2, pixelIdx, rtLaunchIndex, );
rtDeclareVariable(HitRecord, hit, attribute hit, );
rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(Lambertian, brdf, , );
rtDeclareVariable(ShadowPayload, shadowPayload, rtPayload, );
rtDeclareVariable(RadiancePayload, radiancePayload, rtPayload, );
rtDeclareVariable(rtObject, root, , );
rtBuffer<Light*> lights;



RT_FUNCTION float DirectLight(const ONB &onb, const float3 &woW, const float3 &wo) {

	float3 color = make_float3(0.0f);
	unsigned int nLights = lights.size();
	Random2D sampler(&radiancePayload.rng, nLights);
	for (unsigned int i = 0; i < nLights; i++) {
		Light* light = lights[i];
		float2 uniformSample;
		sampler.Next2D(&uniformSample);
		float3 wiW, L;
		float pdf, tMax;
		CALL_LIGHT_VIRTUAL_FUNCTION(Le, = , light, Sample, hit.position, uniformSample, wiW, pdf, tMax);
		ShadowPayload shadowPayload;
		shadowPayload.attenuation = 1.0f;
		Ray shadowRay = make_Ray(hit.position, wiW, RayType::SHADOW, 0.01, tMax);
		rtTrace(root, shadowRay, shadowPayload);
		if (shadowPayload.attenuation > 0.0f) {
			float3 wi = onb.WorldToLocal(wiW);
			float3 BRDF = brdf.Eval(wo, wi);
			float nDotWi = fmaxf(wi.z, 0.0f);
			color += BRDF * nDotWi * Le / pdf;
		}
	}
	color /= (float)nLights;
	return color;
}


RT_FUNCTION float IndirectLight(const ONB &onb, const float3 &woW, const float3 &wo) {

	Random2D sampler(&radiancePayload.rng, 1);
	float2 uniformSample;
	sampler.Next2D(&uniformSample);
	float3 wi;
	float pdf;
	float3 BRDF = brdf.Sample(wo, &wi, &pdf, uniformSample);
	float3 wiW = onb.LocalToWorld(wi);
	Ray radianceRay = make_Ray(hit.position, wiW, RayType::RADIANCE, 0, RT_DEFAULT_MAX);
	RadiancePayload newRadiancePayload;
	newRadiancePayload.depth = radiancePayload.depth + 1;
	newRadiancePayload.rng = radiancePayload.rng;
	rtTrace(root, radianceRay, newRadiancePayload);
	float nDotWi = fmaxf(wi.z, 0.0f);
	color += BRDF * nDotWi * newRadiancePayload.color / pdf;
	
}


RT_PROGRAM void closestHit() {

	ONB onb(hit.normal);
	float3 woW = -ray.direction;
	float3 wo = onb.WorldToLocal(woW);
	float3 color = make_float3(0.0f);
	color += DirectLight(onb, woW, wo);
	if (radiancePayload.depth < MAX_DEPTH) {
		color += IndirectLight(onb, woW, wo);
	}
	radiancePayload.color = color;
}

RT_PROGRAM void anyHit() {

   shadowPayload.attenuation = 0.0f;
   rtTerminateRay();

}

RT_PROGRAM void miss() {

   float3 BLACK = make_float3(0.0f);
   radiancePayload.color = BLACK;

}

