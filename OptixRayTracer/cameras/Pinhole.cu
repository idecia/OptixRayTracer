#include "hip/hip_runtime.h"

#include <optix_world.h>

#include "core/optix_global.h"
#include "core/Ray.h"
#include "core/Payload.h"
#include "cameras/Pinhole.h"
#include "films/Film.h"
#include "core/RNG.h"
#include "samplers/Random2D.h"
using namespace optix;

rtBuffer<RNG, 2> rngs;
rtDeclareVariable(uint2, pixelIdx, rtLaunchIndex, );
rtDeclareVariable(Pinhole, camera, , );
rtDeclareVariable(Film, film, , );
rtDeclareVariable(rtObject, root, , ); 



RT_PROGRAM void pinhole(void) {

	RadiancePayload radiancePayload;
	radiancePayload.color = make_float3(0.0f);
	radiancePayload.depth = 0;
	radiancePayload.rng = rngs[pixelIdx];

	Random2D sampler(&radiancePayload.rng, 1u);
	float2 uniformSample;

	while (sampler.Next2D(&uniformSample)) {

		float2 filmSample = film.Sample(pixelIdx, uniformSample);
		Ray ray = camera.GenerateRay(filmSample);
		rtTrace(root, ray, radiancePayload);
		film.PutSample(filmSample, radiancePayload.color);

	}

	rngs[pixelIdx] = radiancePayload.rng;

}