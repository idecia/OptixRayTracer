#include "hip/hip_runtime.h"

#include <optix_world.h>

#include "core/optix_global.h"
#include "core/Ray.h"
#include "core/Payload.h"
#include "samplers/Util.h"
#include "samplers/RNG.h"
#include "core/ONB.h"
#include "core/math.h"
#include "samplers/Random2D.h"
#include "skyes/Beckers288.h"
#include "lights/EnvironmentLight.h"
#include <optix_device.h>

rtBuffer<RNG> rngs;
//rtBuffer<float2> u;
rtBuffer<float> coeff;
//rtBuffer<float3> coeff;
//rtBuffer<double3> coeff;
rtDeclareVariable(int, Ntot, , );
rtDeclareVariable(int, nSamples, , );
rtDeclareVariable(uint, pixelIdx, rtLaunchIndex, );
rtDeclareVariable(float3, sensorNormal, , );
rtDeclareVariable(float3, point, , );
rtDeclareVariable(float3, sensorPos, , );
rtDeclareVariable(rtObject, buildingWindows, , );
rtDeclareVariable(unsigned int, NskyPatches, , );
rtDeclareVariable(EnvironmentLight, light, , );


RT_PROGRAM void sensor(void) {


	BeckersPayload pl;
	pl.rng = rngs[pixelIdx];

	Random2D sampler(&pl.rng, nSamples);
	float2 unifSample; //= u[pixelIdx];

	while (sampler.Next2D(&unifSample)) {

		//pl.value = make_float3(0.0f);
		pl.value = make_float3(0.0f);
		pl.depth = 0;


		ONB onb(sensorNormal);
		float3 dir = CosineHemisphereSample(unifSample.x, unifSample.y);
		float3 dirW = onb.LocalToWorld(dir);
		//float3 sensorPos2 = make_float3(1.118916, 4.990000, 3.026103);
		//float3 sensorPos2 = make_float3(-1.444052f, -0.000000f, 3.0005f);
		//float3 sensorPos2 = make_float3(-1.490000f, -0.540149f, 0.997289f);
		//dirW = make_float3(0.945326f, 0.091734f, -0.312960f);
		//  dirW = make_float3( -0.456876, -0.88952, -0.004548);
		//sensorPos2 = sensorPos2 - dirW;
		Ray ray = make_Ray(sensorPos, dirW, RayTypeOpt::BECKERS_RADIANCE, 0, RT_DEFAULT_MAX);
		//rtPrintf("%f %f %f\n", dirW.x, dirW.y, dirW.z);

		rtTrace(buildingWindows, ray, pl);

		if (fmaxf(pl.value) > 0.0) {
			float3 value = pl.value * M_PIf / Ntot;
			//float3 value = pl.value;
			//rtPrintf("%d %f %f\n", pl.patchID, (float)value.x, (float)value.x);
			atomicAdd(&coeff[pl.dirID], (float)value.x);
			//atomicAdd(&coeff[pl.dirID].y, (float)value.y);
			//atomicAdd(&coeff[pl.dirID].z, (float)value.z);
			//ns[pl.patchID]++;
		}


	}

	rngs[pixelIdx] = pl.rng;

}


/*
RT_PROGRAM void sensor(void) {

	
	BeckersPayload pl;
	pl.rng = rngs[pixelIdx];

	Random2D sampler(&pl.rng, nSamples);
	float2 unifSample; //= u[pixelIdx];
	while (sampler.Next2D(&unifSample)) {

		
		pl.value = (M_PIf / Ntot) * make_float3(1.0f);
		pl.depth = 0;

		ONB onb(sensorNormal);
		float3 dir = CosineHemisphereSample(unifSample.x, unifSample.y);
		float3 dirW = onb.LocalToWorld(dir);

		Ray ray = make_Ray(sensorPos, dirW, RayTypeOpt::BECKERS_RADIANCE, 0, RT_DEFAULT_MAX);
		//rtPrintf("%f %f %f %f %f %f\n", sensorPos.x, sensorPos.y, sensorPos.z, dirW.x, dirW.y, dirW.z);
		rtTrace(buildingWindows, ray, pl);

	
	
	}

	rngs[pixelIdx] = pl.rng;

}*/

RT_PROGRAM void exception() {

	rtPrintExceptionDetails();

}

/*
RT_PROGRAM void sensor(void) {


	BeckersPayload pl;
	pl.rng = rngs[pixelIdx];

	Random2D sampler(&pl.rng, nSamples);
	float2 unifSample; //= u[pixelIdx];

	while (sampler.Next2D(&unifSample)) {


	
		pl.depth = 0;

		float3 wiW, L;
		float pdf, tMax;

		Random2D sampler(&pl.rng, 1);
		float2 uniformSample;
		sampler.Next2D(&uniformSample);
		L = light.Sample(sensorPos, uniformSample, wiW, pdf, tMax);
		ShadowPayload shadowPayload;
		shadowPayload.attenuation = 1.0f;
		Ray shadowRay = make_Ray(sensorPos, wiW, RayTypeOpt::BECKERS_SHADOW, 0.0, tMax-0.1);
		rtTrace(buildingWindows, shadowRay, shadowPayload);
		if (shadowPayload.attenuation > 0.0f) {
			ONB o(-light.parallelogram.NormalAt(make_float3(1.0f)));
			float3 v = o.WorldToLocal(wiW);
			int dirID = beckers(o.WorldToLocal(wiW));
			float cos = dot(wiW, sensorNormal) > 0 ? dot(wiW, sensorNormal) : 0.0f;
			float value = cos/ pdf;
			value = value / Ntot;
			atomicAdd(&coeff[dirID].x, (float)value);
			atomicAdd(&coeff[dirID].y, (float)value);
			atomicAdd(&coeff[dirID].z, (float)value);
		}

	}
	rngs[pixelIdx] = pl.rng;

}*/
