#include "hip/hip_runtime.h"

#include <optix_world.h>

#include "core/optix_global.h"
#include "core/Ray.h"
#include "core/Payload.h"
#include "cameras/Pinhole.h"
#include "films/Film.h"
#include "core/RNG.h"
#include "samplers/Random2D.h"
#include <optix_device.h>

rtBuffer<RNG, 2> rngs;
rtDeclareVariable(uint2, pixelIdx, rtLaunchIndex, );
rtDeclareVariable(Pinhole, camera, , );
rtDeclareVariable(Film, film, , );
rtDeclareVariable(rtObject, root, , ); 


RT_PROGRAM void pinhole(void) {
	int nSamples = 1000;
	

	RadiancePayload radiancePayload;
	radiancePayload.rng = rngs[pixelIdx];

	Random2D sampler(&radiancePayload.rng, nSamples);
	float2 uniformSample;

	float3 result = make_float3(0.0f);

	while (sampler.Next2D(&uniformSample)) {
		
		radiancePayload.color = make_float3(0.0f);
		radiancePayload.depth = 0;

		float2 filmSample = film.Sample(pixelIdx, uniformSample);
		//rtPrintf("(%d, %d)   ", pixelIdx.x, pixelIdx.y);
		//rtPrintf("(%f, %f)   ", uniformSample.x, uniformSample.y);
		//rtPrintf("(%f, %f)\n", filmSample.x, filmSample.y);
		Ray ray = camera.GenerateRay(filmSample);
		rtTrace(root, ray, radiancePayload);
		//rtPrintf("(%f, %f, %f)\n", radiancePayload.color.x, radiancePayload.color.y, radiancePayload.color.z);
		result += radiancePayload.color;

	}

	film.PutSample(pixelIdx, result / (float)nSamples);
	rngs[pixelIdx] = radiancePayload.rng;


}


RT_PROGRAM void exception() {

	rtPrintExceptionDetails();

}
