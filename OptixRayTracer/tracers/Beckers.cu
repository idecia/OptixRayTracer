#include "hip/hip_runtime.h"
#include "skyes/Beckers288.h"
#include "core/Ray.h"
#include "samplers/Random2D.h"
#include "core/Payload.h"
#include "bxdfs/Lambertian.h"
#include "lights/EnvironmentLight.h"
#include "core/optix_global.h"
#include "core/ONB.h"
#include "core/math.h"
#include <optix_device.h>
#include <hiprand/hiprand_kernel.h>


#define MAX_DEPTH 5


rtDeclareVariable(uint, pixelIdx, rtLaunchIndex, );
rtDeclareVariable(HitRecord, hit, attribute hit, );
rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(Lambertian, brdf, , );
rtDeclareVariable(float3, point, , );
rtDeclareVariable(float3, sensorNormal, , );
rtDeclareVariable(BeckersPayload, beckersPayload, rtPayload, );
rtDeclareVariable(ShadowPayload, shadowPayload, rtPayload, );
rtDeclareVariable(rtObject, buildingWindows, , );
rtDeclareVariable(EnvironmentLight, light, , );
//rtBuffer<float3> coeff;
rtBuffer<float> coeff;
rtDeclareVariable(float3, Le, , );

/*
RT_PROGRAM void closestHit() {
double3 value = make_double3(0.0, 0.0, 0.0);
ONB onb(hit.normal);
float3 woW = -ray.direction;
float3 wo = onb.WorldToLocal(woW);
if (BeckersPayload.depth < MAX_DEPTH) {
Random2D sampler(&BeckersPayload.rng, 1);
float2 uniformSample;
sampler.Next2D(&uniformSample);
float3 wi;
float pdf;
//float3 BRDF = brdf.Sample(wo, &wi, &pdf, uniformSample);
float3 wiW = onb.LocalToWorld(wi);
double3 BRDF = ToDouble(brdf.Sample(wo, &wi, &pdf, uniformSample));
double3 wiWd = ToDouble(wiW);
Ray radianceRay = make_Ray(hit.position + 0.0001*hit.normal, wiW, RayType::RADIANCE, 0, RT_DEFAULT_MAX);
BeckersPayload newBeckersPayload;
newBeckersPayload.depth = BeckersPayload.depth + 1;
newBeckersPayload.rng = BeckersPayload.rng;
rtTrace(root, radianceRay, newBeckersPayload);
//float nDotWi = fabsf(wi.z);
//float nDotWi = fabsf(dot(wiW,hit.normal));
double3 normal = ToDouble(hit.normal);
double nDotWi = fabs(dot(wiWd, normal));
//value = BRDF * nDotWi * newBeckersPayload.value / pdf;
value = BRDF * nDotWi * newBeckersPayload.value / (double)pdf;
BeckersPayload.patchID = newBeckersPayload.patchID;
//rtPrintf("%f %f %f\n", BRDF.x, BRDF.y, BRDF.z);
}
BeckersPayload.value = value;
}
*/

/*
RT_PROGRAM void closestHit() {
}
*/

/*
RT_PROGRAM void closestHit() {

	//rtPrintf("%DIRECT LIGHT entro %f\n", beckersPayload.value.x);
	ONB onb(hit.normal);
	ONB onbWindow(-hit.normal);
	float3 woW = -ray.direction;
	float3 wo = onb.WorldToLocal(woW);


	if ((beckersPayload.depth == 0) && (Le.x > 0)) {
		//rtPrintf("%HIT LIGHT %f %f %f\n", hit.position.x, hit.position.y, hit.position.z);
		float3 value = beckersPayload.value;
		//float3 value = make_float3(1.0f);
		int dirID = beckers(onbWindow.WorldToLocal(-woW));
		atomicAdd(&coeff[dirID], (float)value.x);
		//atomicAdd(&coeff[dirID].x, (float)value.x);
		//atomicAdd(&coeff[dirID].y, (float)value.y);
		//atomicAdd(&coeff[dirID].z, (float)value.z);
		return;
	}
	if ((Le.x > 0))
		return;

	//direct light
	unsigned int nLights = 1;
	int nSamples = 1;
	Random2D sampler(&beckersPayload.rng, nSamples);
	for (unsigned int i = 0; i < nLights; i++) {
		float2 uniformSample;
		sampler.Next2D(&uniformSample);
		float3 wiW, L;
		float pdf, tMax;
		L = light.Sample(hit.position, uniformSample, wiW, pdf, tMax);
		ShadowPayload shadowPayload;
		shadowPayload.attenuation = 1.0f;
		Ray shadowRay = make_Ray(hit.position, wiW, RayTypeOpt::BECKERS_SHADOW, 0.0, tMax - 0.0001);
		rtTrace(buildingWindows, shadowRay, shadowPayload);
		if (shadowPayload.attenuation > 0.0f) {
			//rtPrintf("%DIRECT LIGHT %f %f %f %f %f\n", L.x, pdf, wiW.x, wiW.y, wiW.z);
			float3 wi = onb.WorldToLocal(wiW);
			float3 BRDF = brdf.Eval(wo, wi);

			float nDotWi = fmaxf(wi.z, 0.0f);

			float3 value = beckersPayload.value * BRDF * nDotWi * L / pdf;
			ONB o(-light.parallelogram.NormalAt(make_float3(1.0f)));
			float3 v = o.WorldToLocal(wiW);
			int dirID = beckers(o.WorldToLocal(wiW));
			//if (pdf == 0)
			//rtPrintf("%DIRECT LIGHT entro %f %f %f %f %f %f\n", value.x, BRDF.x, nDotWi, L.x, pdf, coeff[dirID].x);
			atomicAdd(&coeff[dirID], (float)value.x);
			//atomicAdd(&coeff[dirID].x, (float)value.x);
			//atomicAdd(&coeff[dirID].y, (float)value.y);
			//atomicAdd(&coeff[dirID].z, (float)value.z);
		}
	}

	if (beckersPayload.depth < MAX_DEPTH) {
		Random2D sampler(&beckersPayload.rng, 1);
		float2 uniformSample;
		sampler.Next2D(&uniformSample);


		float3 wi;
		float pdf;
		float3 BRDF;

		BRDF = brdf.Sample(wo, &wi, &pdf, uniformSample);
		//	rtPrintf("%f \n", brdf.getR().x);

		float3 wiW = onb.LocalToWorld(wi);
		Ray radianceRay;
		radianceRay = make_Ray(hit.position + 0.00001*hit.normal, wiW, RayTypeOpt::BECKERS_RADIANCE, 0, RT_DEFAULT_MAX);
		float nDotWi = fabsf(dot(wiW, hit.normal));
		BeckersPayload newBeckersPayload;
		newBeckersPayload.depth = beckersPayload.depth + 1;
		newBeckersPayload.rng = beckersPayload.rng;
		//	if (pdf == 0)
		//rtPrintf("%DIRECT LIGHT entro %f %f %f %f %f %f %f %f\n", beckersPayload.value.x, BRDF.x, nDotWi, pdf, uniformSample.x, uniformSample.y, u1, u2);
		newBeckersPayload.value = beckersPayload.value * BRDF * nDotWi / pdf;
		rtTrace(buildingWindows, radianceRay, newBeckersPayload);
	}
}*/


RT_PROGRAM void closestHit() {

	ONB onb(hit.normal);
	ONB onbWindow(-hit.normal);
	float3 woW = -ray.direction;
	float3 wo = onb.WorldToLocal(woW);
	if ((Le.x > 0) ) {
		beckersPayload.value = make_float3(1.0);
		beckersPayload.dirID = beckers(onbWindow.WorldToLocal(-woW));
		//rtPrintf("%f  %f %f \n", ray.direction.x, ray.direction.y, ray.direction.z);
		//rtPrintf("%d \n", beckersPayload.depth);
		return;
	}

	float3 value = make_float3(0.0f);
	if (beckersPayload.depth < MAX_DEPTH) {
		Random2D sampler(&beckersPayload.rng, 1);
		float2 uniformSample;
		sampler.Next2D(&uniformSample);
		float3 wi;
		float pdf;
		float3 BRDF;
		BRDF = brdf.Sample(wo, &wi, &pdf, uniformSample);
		//	rtPrintf("%f \n", brdf.getR().x);
		float3 wiW = onb.LocalToWorld(wi);
		Ray radianceRay;
		radianceRay = make_Ray(hit.position + 0.0*hit.normal, wiW, RayTypeOpt::BECKERS_RADIANCE, 0.000001, RT_DEFAULT_MAX);
		BeckersPayload newBeckersPayload;
		newBeckersPayload.depth = beckersPayload.depth + 1;
		newBeckersPayload.rng = beckersPayload.rng;
		rtTrace(buildingWindows, radianceRay, newBeckersPayload);
		//float nDotWi = fabsf(wi.z);
		//float nDotWi = fabsf(dot(wiW, hit.normal));
		float nDotWi = fmaxf(wi.z, 0.0f);
		value = BRDF * nDotWi * newBeckersPayload.value / pdf;
		beckersPayload.dirID = newBeckersPayload.dirID;
		//rtPrintf("%f %f %f\n", BRDF.x, BRDF.y, BRDF.z);
	}
	beckersPayload.value = value;
}


RT_PROGRAM void anyHit() {

	shadowPayload.attenuation = 0.0f;
	rtTerminateRay();

}

RT_PROGRAM void miss() {

	//BeckersPayload.value = make_double3(1.0,1.0,1.0);

	beckersPayload.value = make_float3(1.0);
	beckersPayload.dirID = 0;
	//if (beckersPayload.dirID == 0) {
	//float t = -ray.origin.y / ray.direction.y;
	//rtPrintf(" %f %f %f %f %f %f \n", (ray.origin + t* ray.direction).x, (ray.origin + t* ray.direction).y, (ray.origin + t* ray.direction).z, ray.direction.x, ray.direction.y, ray.direction.z);

	//rtPrintf("%d %f %f %f %f %f %f \n", BeckersPayload.depth, ray.origin.x, ray.origin.y, ray.origin.z, ray.direction.x, ray.direction.y, ray.direction.z);

	//rtPrintf(" %f %f %f    %f %f %f\n", ray.origin.x, ray.origin.y, ray.origin.z, (ray.origin + 10 * ray.direction).x, (ray.origin + 10 * ray.direction).y, (ray.origin + 10 * ray.direction).z);

	//	rtPrintf("(%f %f %f)  (%f %f %f) %d %f \n   ", ray.origin.x, ray.origin.y, ray.origin.z, ray.direction.x, ray.direction.y, ray.direction.z, BeckersPayload.patchID, BeckersPayload.value.x);
	//rtPrintf("%d\n",  BeckersPayload.patchID);

		//rtPrintf("%f %f %f    \n   ", ray.origin.x, ray.origin.y, ray.origin.z );
	//}
}
