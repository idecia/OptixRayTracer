#include "hip/hip_runtime.h"
#include "optix_global.h"
#include "Camera.h"
#include "Film.h"


using namespace optix;

rtBuffer<float3, 2>   outputBuffer;
rtBuffer<RNG, 2> rngs;
rtDeclareVariable(uint2, pixelIdx, rtLaunchIndex, );
rtDeclareVariable(Camera, camera, , );
rtDeclareVariable(Film, film, , );
rtDeclareVariable(rtObject, root, , );

struct Payload {

	RNG rng;
	float3 color;

};

RT_PROGRAM void pinhole(void) {

	Payload payload;
	payload.rng = rngs[pixelIdx];
	Random2D sampler(&payload.rng, 1u);
	float2 uniformSample;
	while (sampler.Next2D(&uniformSample)) {
		float2 filmSample = film.Sample(pixelIdx, uniformSample);
		Ray ray = camera.GenerateRay(filmSample);
		rtTrace(root, ray, payload);
		film.PutSample(filmSample, payload.color)
	}
	rngs[pixelIdx] = payload.rng;

}