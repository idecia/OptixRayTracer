#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include "Camera.h"
#include "Film.h"


using namespace optix;

rtBuffer<float3, 2>   outputBuffer;
rtDeclareVariable(uint2, index, rtLaunchIndex, );
rtDeclareVariable(Camera, camera, , );
rtDeclareVariable(Film, film, , );
rtDeclareVariable(rtObject, root, , );

struct Payload {

	float3 color;

};

RT_PROGRAM void pinhole(void) {

	float2 sample = //TODO
		Ray ray = camera.generateRay(sample);
	Payload payload;
	rtTrace(root, ray, payload);
	film.put(sample, payload.color)

}