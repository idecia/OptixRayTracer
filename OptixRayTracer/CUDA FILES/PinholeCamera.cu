#include "hip/hip_runtime.h"

#include "Camera.h"
#include "ViewPlane.h"

	
#include <optix.h>
#include <optixu/optixu_math_namespace.h>


using namespace optix;

rtBuffer<float3, 2>   outputBuffer;
rtDeclareVariable(uint2, index, rtLaunchIndex, );
rtDeclareVariable(Camera, camera, , );
rtDeclareVariable(ViewPlane, viewPlane, , );
rtDeclareVariable(rtObject, topObject, , );

struct Payload {

	float3 color;

};

RT_PROGRAM void pinholeCamera(void) {

	float2 sample = samplePixel(viewPlane, index.x , index.y);
	Ray ray = generateRay(camera, sample   );
	Payload payload;
	rtTrace(topObject, ray, payload);
	outputBuffer[index] = payload.color;

}