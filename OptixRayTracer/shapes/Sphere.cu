#include "hip/hip_runtime.h"
#include "core/optix_global.h"
#include "core/Ray.h"
#include "shapes/Sphere.h"

#include <optix_world.h>

using namespace optix;


rtDeclareVariable(Sphere, sphere, , );
rtDeclareVariable(HitRecord, hit, attribute hit, );
rtDeclareVariable(Ray, ray, rtCurrentRay, );

RT_FUNCTION void setHitRecord(float t) {

	hit.position = ray.origin + t * ray.direction;
	hit.normal = sphere.NormalAt(hit.position);

}

RT_PROGRAM void intersect(int primIdx) {
		
	float3 center = sphere.GetCenter();
	float radius  = sphere.GetRadius();
	float3 O = ray.origin - center;
	float b = dot(O, ray.direction);
	float c = dot(O, O) - radius * radius;
	float discriminant = b * b - c;
	if (discriminant > 0.0f) {
		float sqrtDiscriminant = sqrtf(discriminant);
		float tMin = (-b - sqrtDiscriminant);
		bool checkSecond = true;
		if (rtPotentialIntersection(tMin)) {
			setHitRecord(tMin);
			if (rtReportIntersection(0)) 
				checkSecond = false;
		}
		if (checkSecond) {
			float tMax = (-b + sqrtDiscriminant);
			if (rtPotentialIntersection(tMax)) {
				setHitRecord(tMax);
				rtReportIntersection(0);
			}
		}
	}
}

RT_PROGRAM void boundingBox(int, float result[6]) {

	float3 center = sphere.GetCenter();
    float3 radius3f = make_float3(sphere.GetRadius());
	optix::Aabb* aabb = (optix::Aabb*) result;
    aabb->m_min = center - radius3f;
    aabb->m_max = center + radius3f;
}
