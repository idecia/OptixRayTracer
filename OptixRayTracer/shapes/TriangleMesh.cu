#include "hip/hip_runtime.h"
#include "core/optix_global.h"
#include "core/Ray.h"
#include "shapes/Triangle.h"
#include "core/math.h"
#include <optix_world.h>

using namespace optix;


rtBuffer<float3> vertexBuffer;
rtBuffer<float3> normalBuffer;
rtBuffer<int3> indexBuffer;
rtDeclareVariable(HitRecord, hit, attribute hit, );
rtDeclareVariable(Ray, ray, rtCurrentRay, );
//rtDeclareVariable(float3, geometricNormal, attribute geometricNormal, );
//rtDeclareVariable(float3, shadingNormal, attribute shadingNormal, );


RT_PROGRAM void intersect(int primIdx) {

	int3 index = indexBuffer[primIdx];

	float3 p0 = vertexBuffer[index.x];
	float3 p1 = vertexBuffer[index.y];
	float3 p2 = vertexBuffer[index.z];

	float3 E1 = p1 - p0;
	float3 E2 = p2 - p0;

	float3 S1 = cross(ray.direction, E2);
	float invDivisor = 1.0f / dot(S1, E1);

	float3 T = ray.origin - p0;
	float u = dot(T, S1) * invDivisor;

	float3 S2 = cross(T, E1);
	float v = dot(ray.direction, S2) * invDivisor;

	if (u < 0.0f || v < 0.0f || u + v > 1.0f) return;

	float t = dot(E2, S2) * invDivisor;
	if (rtPotentialIntersection(t)) {
		float3 p = ray.origin + ray.direction * t;
		p = p0 + u*E1 + v*E2;
		float3 normal = normalize(cross(E1, E2));
		//rtPrintf("(%f, %f, %f)\n", normal.x, normal.y, normal.z);
		hit.position = p;
		hit.normal = normal;
		rtReportIntersection(0);

	}

}


RT_PROGRAM void boundingBox(int primIdx, float result[6]) {

	const int3 index = indexBuffer[primIdx];

	const float3 p0 = vertexBuffer[index.x];
	const float3 p1 = vertexBuffer[index.y];
	const float3 p2 = vertexBuffer[index.z];

	optix::Aabb* aabb = (optix::Aabb*) result;
	aabb->m_min = fminf(fminf(p0, p1), p2);
	aabb->m_max = fmaxf(fmaxf(p0, p1), p2);

}



