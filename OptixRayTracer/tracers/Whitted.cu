#include "hip/hip_runtime.h"
#include "lights/PointLight.h"
#include "core/Ray.h"
#include "cameras/Pinhole"
#include "core/optix_global.h"


rtDeclareVariable(uint2, pixelIdx, rtLaunchIndex, );
rtDeclareVariable(Pinhole, camera, , )
rtDeclareVariable(HitRecord, hit, , )
rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(Lambertian, brdf, , );
rtDeclareVariable(ShadowPayload, shadowPayload, rtPayload, );
rtDeclareVariable(RadiancePayload, radiancePayload, rtPayload, );


rtBuffer<PointLight>     lights;

RT_PROGRAM void closestHit() {

	//float3 color = make_float3(0.0f);
	float3 color = brdf.Eval(NULL) * 0.1; //simulate ambient light
	unsigned int numLights = lights.size();
	for (unsigned int i = 0; i < numLights; i++) {

		PointLight light = lights[i];
		float3 L = light.getPosition() - hit.position;
		float LDist = length(L);
		L = normalize(L);
		ShadowPayload shadowPayload;
		shadowPayload.attenuation = 1.0f;
		Ray shadowRay = make_Ray(hit.position, L, RayType::RADIANCE, shadowPayload, 0.05, LDist);
		rtTrace(root, shadowRay, shadowPaylod);
		if (shadowPayload.attenuation > 0.0f) {
			float nDotl = dot(hit.normal, L);
			float BRDF = brdf.Eval(NULL);
			color += BRDF * nDotl * light.getColor();
		}
	}
	radiancePayload.color = color;
	
}

RT_PROGRAM void anyHit() {

	shadowPayload.attenuation = 0.0f;
	rtTerminateRay();

}


RT_PROGRAM void miss() {

	float3 BLACK = make_float3(0.0f);
	radiancePayload.color = BLACK;

}

