#include "hip/hip_runtime.h"

#include <optix_world.h>

#include "core/optix_global.h"
#include "core/Ray.h"
#include "core/Payload.h"
#include "samplers/Util.h"
#include "samplers/RNG.h"
#include "core/ONB.h"
#include "core/math.h"
#include "skyes/Beckers288.h"
#include "samplers/Random2D.h"
#include <optix_device.h>

rtBuffer<RNG> rngs;
rtBuffer<float3, 2> env ;
rtDeclareVariable(uint, pixelIdx, rtLaunchIndex, );
rtDeclareVariable(float3, sensorNormal, , );
rtDeclareVariable(float3, sensorPos, , );
rtDeclareVariable(rtObject, root, , );
rtDeclareVariable(int, Ntot, , );
rtDeclareVariable(int, nSamples , , );

RT_PROGRAM void sensor(void) {
	//rtPrintf("ok");
	ReinhartPayload pl;
	pl.rng = rngs[pixelIdx];

	Random2D sampler(&pl.rng, nSamples);
	float2 unifSample; 

	while (sampler.Next2D(&unifSample)) {
		pl.value = make_float3(0.0f);
		pl.depth = 0;

		ONB onb(sensorNormal);
		float3 dir = UniformHemisphereSample(unifSample.x, unifSample.y);
		float3 dirW = onb.LocalToWorld(dir);
		Ray ray = make_Ray(sensorPos, dirW, RayTypeOpt::REINHART_RADIANCE, 0, RT_DEFAULT_MAX);
		rtTrace(root, ray, pl);
		//rtPrintf("%f %f %f %f %f %f\n", dir.x, dir.y, dir.z, sensorPos.x, sensorPos.y, sensorPos.z);
		if (fmaxf(pl.value) > 0.0) {
			uint2 index;
			index.x = beckers(dir);
			index.y = pl.patchID;
			//if (index.x == 242) {
				//rtPrintf("- %f %f %f %f %d %d\n", dir.x, dir.y, dir.z, env[index].x, beckers(dir), index.y);
			//}
			//ncell = 288; AREGLAR ESTO QUE ESTA HARCODEADO!
			//float3 value = (288/2*M_PIf) * 2*M_PIf*pl.value/Ntot;
			float3 value = (288 * pl.value) / Ntot;
			//float3 value =pl.value ;
			atomicAdd(&env[index].x, (float)value.x);
			atomicAdd(&env[index].y, (float)value.y);
			atomicAdd(&env[index].z, (float)value.z); // falta multiplicar por 2pi/N
		}
		



	}
	rngs[pixelIdx] = pl.rng;

	/*for (int i = 0; i < 288; i++)  {
		for (int j = 0; j < 146; j++) {
			uint2 index;
			index.x = i;
			index.y = j;
			env[index] = make_float3(j);
			//rtPrintf("%f    ", v.x);
		}
		//rtPrintf("\n");
	}*/

}


RT_PROGRAM void exception() {

	rtPrintExceptionDetails();

}
