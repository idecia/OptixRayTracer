#include "hip/hip_runtime.h"
#include "lights/AreaLight.h"
#include "core/Ray.h"
#include "cameras/Pinhole.h"
#include "samplers\Random2D.h"
#include "core/Payload.h"
#include "brdfs/Lambertian.h"
#include "core/optix_global.h"
#include <optix_device.h>


rtDeclareVariable(uint2, pixelIdx, rtLaunchIndex, );
rtDeclareVariable(Pinhole, camera, , );
rtDeclareVariable(HitRecord, hit, attribute hit, );
rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(Lambertian, brdf, , );
rtDeclareVariable(ShadowPayload, shadowPayload, rtPayload, );
rtDeclareVariable(RadiancePayload, radiancePayload, rtPayload, );
rtDeclareVariable(rtObject, root, , );
rtBuffer<AreaLight> lights;

RT_PROGRAM void closestHit() {

	//float3 color = make_float3(0.0f);
	BRDFQueryRecord  dummy;

	//ambient light
	float3 color = brdf.Eval(dummy) * 0.1; 

	//direct light
	unsigned int numLights = lights.size();
	int nSamples = 1;
	Random2D sampler(&radiancePayload.rng, nSamples);
	for (unsigned int i = 0; i < numLights; i++) {
		AreaLight light = lights[i];
		float2 uniformSample;
		while (sampler.Next2D(&uniformSample)) {
			float3 L = light.sample(uniformSample) - hit.position;
			float LDist = length(L);
			L = normalize(L);
			ShadowPayload shadowPayload;
			shadowPayload.attenuation = 1.0f;
			Ray shadowRay = make_Ray(hit.position, L, RayType::SHADOW, 0.6, LDist);
			rtTrace(root, shadowRay, shadowPayload);
			if (shadowPayload.attenuation > 0.0f) {
				float nDotl = dot(hit.normal, L);
				float3 BRDF = brdf.Eval(dummy);
				color += BRDF * nDotl * light.GetColor();
			}
		}
		
	}
	color /= (float)nSamples;
	
	radiancePayload.color = color;
	

}

RT_PROGRAM void anyHit() {

	shadowPayload.attenuation = 0.0f;
	rtTerminateRay();

}

RT_PROGRAM void miss() {

	float3 BLACK = make_float3(0.0f);
	radiancePayload.color = BLACK;

}

