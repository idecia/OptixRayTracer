#include "hip/hip_runtime.h"
#include "lights/AllLights.h"
#include "core/Ray.h"
#include "samplers/Random2D.h"
#include "core/Payload.h"
#include "bxdfs/Lambertian.h"
#include "core/optix_global.h"
#include "core/ONB.h"
#include <optix_device.h>

#define MAX_DEPTH 4

rtDeclareVariable(uint2, pixelIdx, rtLaunchIndex, );
rtDeclareVariable(HitRecord, hit, attribute hit, );
rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(Lambertian, brdf, , );
rtDeclareVariable(float3, Le, , );
rtDeclareVariable(ShadowPayload, shadowPayload, rtPayload, );
rtDeclareVariable(RadiancePayload, radiancePayload, rtPayload, );
rtDeclareVariable(rtObject, root, , );
rtBuffer<Light*> lights;



RT_PROGRAM void closestHit() {
	if ((radiancePayload.depth == 0) && (Le.x > 0)){
		radiancePayload.color = Le;
		return;
	}
   float3 color = make_float3(0.0f);
   ONB onb(hit.normal);
   float3 woW = -ray.direction;
   float3 wo  = onb.WorldToLocal(woW);
 
   //direct light
   unsigned int nLights = lights.size();
   int nSamples = nLights;
   Random2D sampler(&radiancePayload.rng, nSamples);
   for (unsigned int i = 0; i < nLights; i++) {
      Light* light = lights[i];
      float2 uniformSample;
	  sampler.Next2D(&uniformSample);
      float3 wiW, L; 
      float pdf, tMax;
	  CALL_LIGHT_VIRTUAL_FUNCTION(L, = , light, Sample, hit.position, uniformSample, wiW, pdf, tMax);
      ShadowPayload shadowPayload;
      shadowPayload.attenuation = 1.0f;
      Ray shadowRay = make_Ray(hit.position, wiW, RayType::SHADOW, 0.01, tMax);
      rtTrace(root, shadowRay, shadowPayload);
      if (shadowPayload.attenuation > 0.0f) {
		 float3 wi   = onb.WorldToLocal(wiW);
         float3 BRDF = brdf.Eval(wo, wi);
         float nDotWi = fmaxf(wi.z, 0.0f);
		 color += BRDF * nDotWi * L / pdf;
		// rtPrintf("nLights (%d)\n", light->GetType());
		// rtPrintf("light (%f, %f, %f)\n", L.x, L.y, L.z);
		 //rtPrintf("color (%f, %f, %f)\n", color.x, color.y, color.z);
       }  
   }
   color /= (float)nSamples;

   //indirect light
   if (radiancePayload.depth < MAX_DEPTH) {
	  Random2D sampler(&radiancePayload.rng, 1);
	  float2 uniformSample;
	  sampler.Next2D(&uniformSample);
	  float3 wi; 
	  float pdf;
	  float3 BRDF = brdf.Sample(wo, &wi, &pdf, uniformSample);
	  float3 wiW  = onb.LocalToWorld(wi);
      Ray radianceRay = make_Ray(hit.position, wiW, RayType::RADIANCE, 0, RT_DEFAULT_MAX);
	  RadiancePayload newRadiancePayload;
	  newRadiancePayload.depth = radiancePayload.depth + 1;
	  newRadiancePayload.rng   = radiancePayload.rng;
	  rtTrace(root, radianceRay, newRadiancePayload);
	    float nDotWi = fmaxf(wi.z, 0.0f);
	  color += BRDF * nDotWi * newRadiancePayload.color / pdf;
   } 
   
   radiancePayload.color = color;

   

}

RT_PROGRAM void anyHit() {

   shadowPayload.attenuation = 0.0f;
   rtTerminateRay();

}

RT_PROGRAM void miss() {

   float3 BLACK = make_float3(0.0f);
   radiancePayload.color = BLACK;

}

