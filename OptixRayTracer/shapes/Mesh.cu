#include "hip/hip_runtime.h"
#include "core/optix_global.h"
#include "core/Ray.h"
#include "shapes/Triangle.h"
#include "core/math.h"
#include <optix_world.h>

using namespace optix;


rtBuffer<float3> vertexBuffer;
rtBuffer<float3> normalBuffer;
rtBuffer<int3> indexBuffer;
rtDeclareVariable(HitRecord, hit, attribute hit, );
rtDeclareVariable(Ray, ray, rtCurrentRay, );
//rtDeclareVariable(float3, geometricNormal, attribute geometricNormal, );
//rtDeclareVariable(float3, shadingNormal, attribute shadingNormal, );



RT_PROGRAM void intersect(int primIdx) {

	int3 index = indexBuffer[primIdx];

	float3 p0 = vertexBuffer[index.x];
	float3 p1 = vertexBuffer[index.y];
	float3 p2 = vertexBuffer[index.z];

	float3 E1 = p1 - p0;
	float3 E2 = p2 - p0;

	float3 S1 = cross(ray.direction, E2);
	float invDivisor = 1.0f / dot(S1, E1);

	float3 T = ray.origin - p0;
	float u = dot(T, S1) * invDivisor;

	float3 S2 = cross(T, E1);
	float v = dot(ray.direction, S2) * invDivisor;

	if (u < 0.0f || v < 0.0f || u + v > 1.0f) return;

	float t = dot(E2, S2) * invDivisor;
	if (rtPotentialIntersection(t)) {
		float eps = 0.001;
		t = t < eps ? t : t - eps;
		float3 p = ray.origin + ray.direction * t;
		//p = p0 + u*E1 + v*E2;
		float3 normal = normalize(cross(E1, E2));
		//rtPrintf("(%f, %f, %f)\n", normal.x, normal.y, normal.z);
		hit.position = p;
		hit.normal = normal;
		rtReportIntersection(0);

	}

}


/*
RT_PROGRAM void intersect(int primIdx) {
	
	int3 index = indexBuffer[primIdx];

	double3 p0 = ToDouble(vertexBuffer[index.x]);
	double3 p1 = ToDouble(vertexBuffer[index.y]);
	double3 p2 = ToDouble(vertexBuffer[index.z]);
	
	double3 E1 = p1 - p0;
	double3 E2 = p2 - p0;

	double3 S1 = cross(ToDouble(ray.direction), E2);
	double invDivisor = 1.0 / dot(S1, E1);

	double3 T = ToDouble(ray.origin) - p0;
	double u = dot(T, S1) * invDivisor;

	double3 S2 = cross(T, E1);
	double v = dot(ToDouble(ray.direction), S2) * invDivisor;

	if (u < 0.0 || v < 0.0 || u + v > 1.0) return;

	float t = (float)dot(E2, S2) * invDivisor;
	if (rtPotentialIntersection(t)) {
		float eps = 0.001;
		t = t < eps? t : t - eps;
		double3 p = ToDouble(ray.origin) + ToDouble(ray.direction) * t;
		double3 normal = normalize(cross(E1, E2));
		//rtPrintf("(%f, %f, %f)\n", normal.x, normal.y, normal.z);
		hit.position = ToFloat(p);
		hit.normal = ToFloat(normal);
		rtReportIntersection(0);

	}

}*/

RT_PROGRAM void boundingBox(int primIdx, float result[6]) {

	const int3 index = indexBuffer[primIdx];

	const float3 p0 = vertexBuffer[index.x];
	const float3 p1 = vertexBuffer[index.y];
	const float3 p2 = vertexBuffer[index.z];

	optix::Aabb* aabb = (optix::Aabb*) result;
	aabb->m_min = fminf(fminf(p0, p1), p2);
	aabb->m_max = fmaxf(fmaxf(p0, p1), p2);

}



