#include "hip/hip_runtime.h"

#include <optix_world.h>

#include "core/optix_global.h"
#include "core/Ray.h"
#include "core/Payload.h"
#include "samplers/Util.h"
#include "samplers/RNG.h"
#include "core/ONB.h"
#include "core/math.h"
#include "samplers/Random2D.h"
#include <optix_device.h>

rtBuffer<RNG> rngs;
//rtBuffer<float2> u;
//rtBuffer<float3> coeff;
rtBuffer<float> coeff;
//rtBuffer<double3> coeff;
rtDeclareVariable(double, out, , );
rtDeclareVariable(int, Ntot, ,);
rtDeclareVariable(int, nSamples, , );
rtDeclareVariable(uint, pixelIdx, rtLaunchIndex, );
rtDeclareVariable(float3, sensorNormal, , );
rtDeclareVariable(float3, point, , );
rtDeclareVariable(float3, sensorPos, , );
rtDeclareVariable(rtObject, root, , );
rtDeclareVariable(unsigned int, NskyPatches, , );


RT_PROGRAM void sensor(void) {

	ReinhartPayload pl;
	pl.rng = rngs[pixelIdx];

	Random2D sampler(&pl.rng, nSamples);
	float2 unifSample; //= u[pixelIdx];


	while (sampler.Next2D(&unifSample)) {

		//pl.value = make_float3(0.0f);
		pl.value = make_float3(0.0f);
		pl.depth = 0;


		ONB onb(sensorNormal);
		float3 dir   = CosineHemisphereSample(unifSample.x, unifSample.y);
		float3 dirW  = onb.LocalToWorld(dir);
		//float3 sensorPos2 = make_float3(1.118916, 4.990000, 3.026103);
		//float3 sensorPos2 = make_float3(-1.444052f, -0.000000f, 3.0005f);
		//float3 sensorPos2 = make_float3(-1.490000f, -0.540149f, 0.997289f);
		//dirW = make_float3(0.945326f, 0.091734f, -0.312960f);
	  //  dirW = make_float3( -0.456876, -0.88952, -0.004548);
		//sensorPos2 = sensorPos2 - dirW;
		Ray ray = make_Ray(sensorPos, dirW, RayTypeOpt::REINHART_RADIANCE, 0, RT_DEFAULT_MAX);
		//rtPrintf("%f %f %f\n", dirW.x, dirW.y, dirW.z);

		rtTrace(root, ray, pl);
		
		if (fmaxf(pl.value) > 0.0) {
			float3 value = pl.value * M_PIf/Ntot;
			//float3 value = pl.value/Ntot;
			
			atomicAdd(&coeff[pl.patchID], (float)value.x);
			//atomicAdd(&coeff[pl.patchID].x, (float)value.x);
		//	atomicAdd(&coeff[pl.patchID].y, (float)value.y);
			//atomicAdd(&coeff[pl.patchID].z, (float)value.z);
			//atomicAdd(&coeff[pl.patchID].z, (float)out);
		
			//ns[pl.patchID]++;
		}
		
		
	}

	rngs[pixelIdx] = pl.rng;

}


RT_PROGRAM void exception() {

	rtPrintExceptionDetails();

}


