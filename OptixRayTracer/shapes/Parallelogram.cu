#include "hip/hip_runtime.h"
#include "core/optix_global.h"
#include "core/Ray.h"
#include "shapes/Parallelogram.h"
#include <optix_world.h>

using namespace optix;


rtDeclareVariable(Parallelogram, parallelogram, , );
rtDeclareVariable(HitRecord, hit, attribute hit, );
rtDeclareVariable(Ray, ray, rtCurrentRay, );


RT_PROGRAM void intersect(int primIdx) {

	float4 plane = parallelogram.GetPlane();
	float3 normal = make_float3(plane);

	float nDotD = dot(normal, ray.direction);
	float nDotO = dot(normal, ray.origin);
	float t = (plane.w - nDotO) / nDotD;

	if (t <= ray.tmin || t >= ray.tmax) return;

	float3 p = ray.origin + ray.direction * t;
	float3 p0 = parallelogram.GetP0();
	float3 p1 = parallelogram.GetP1();
	float3 p2 = parallelogram.GetP2();

	float3 side1 = p1 - p0;// normalize(p1 - p0);
	float u = dot(p - p0, side1) / dot (side1, side1);
	if (u < 0 || u > 1) return;


	float3 side2 = p2 - p0;//normalize(p2 - p0);
	float v = dot(p - p0, side2) / dot(side2, side2);
	if (v < 0 || v > 1) return;

	if (rtPotentialIntersection(t)) {
		hit.position = p;
		hit.normal = normal;
		rtReportIntersection(0);
	}


}

RT_PROGRAM void boundingBox(int, float result[6]) {

	float3 p0 = parallelogram.GetP0();
	float3 p1 = parallelogram.GetP1();
	float3 p2 = parallelogram.GetP2();
	float3 p3 = p0 + p1 + p2;

	optix::Aabb* aabb = (optix::Aabb*) result;
	aabb->m_min = fminf(fminf(p0, p1), fminf(p2, p3));
	aabb->m_max = fmaxf(fmaxf(p0, p1), fmaxf(p2, p3));

}
