#include "hip/hip_runtime.h"
#include "skyes/Reinhart145.h"
#include "core/Ray.h"
#include "samplers/Random2D.h"
#include "samplers/Hemisphere.h"
#include "core/Payload.h"
#include "bxdfs/Lambertian.h"
#include "bxdfs/ThinGlass.h"
#include "core/optix_global.h"
#include "core/ONB.h"
#include "core/math.h"
#include <optix_device.h>

#define MAX_DEPTH 6

rtDeclareVariable(uint, pixelIdx, rtLaunchIndex, );
rtDeclareVariable(HitRecord, hit, attribute hit, );
rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(Lambertian, brdf, , );
rtDeclareVariable(uint, glass, , );
rtDeclareVariable(ThinGlass, brdfGlass, , );
rtDeclareVariable(float3, point, , );
rtDeclareVariable(ReinhartPayload, reinhartPayload, rtPayload, );
rtDeclareVariable(ShadowPayload, shadowPayload, rtPayload, );
rtDeclareVariable(rtObject, root, , );
/*
RT_PROGRAM void closestHit() {
	
	double3 value = make_double3(0.0, 0.0, 0.0);
	ONB onb(hit.normal);
	float3 woW = -ray.direction;
	float3 wo = onb.WorldToLocal(woW);

	if (reinhartPayload.depth < MAX_DEPTH) {
		Random2D sampler(&reinhartPayload.rng, 1);
		float2 uniformSample;
		sampler.Next2D(&uniformSample);
		float3 wi;
		float pdf;
		//float3 BRDF = brdf.Sample(wo, &wi, &pdf, uniformSample);
		float3 wiW = onb.LocalToWorld(wi);
		double3 BRDF = ToDouble(brdf.Sample(wo, &wi, &pdf, uniformSample));
		double3 wiWd = ToDouble(wiW);
		Ray radianceRay = make_Ray(hit.position + 0.0001*hit.normal, wiW, RayType::RADIANCE, 0, RT_DEFAULT_MAX);
		ReinhartPayload newReinhartPayload;
		newReinhartPayload.depth = reinhartPayload.depth + 1;
		newReinhartPayload.rng = reinhartPayload.rng;
		rtTrace(root, radianceRay, newReinhartPayload);
		//float nDotWi = fabsf(wi.z);
		//float nDotWi = fabsf(dot(wiW,hit.normal));
		double3 normal = ToDouble(hit.normal);
		double nDotWi = fabs(dot(wiWd, normal));
		//value = BRDF * nDotWi * newReinhartPayload.value / pdf;
		value = BRDF * nDotWi * newReinhartPayload.value / (double)pdf;
		reinhartPayload.patchID = newReinhartPayload.patchID;
		
			//rtPrintf("%f %f %f\n", BRDF.x, BRDF.y, BRDF.z);
	}
	reinhartPayload.value = value;
}
*/

RT_PROGRAM void closestHit() {
	
	//	rtPrintf("Hit: %f %f %f %\n", hit.position.x, hit.position.y, hit.position.z);
	//	return;
	float3 value = make_float3(0.0f);
	ONB onb(hit.normal);
	float3 woW = -ray.direction;
	float3 wo = onb.WorldToLocal(woW);

	if (reinhartPayload.depth < MAX_DEPTH) {
		Random2D sampler(&reinhartPayload.rng, 1);
		float2 uniformSample;
		sampler.Next2D(&uniformSample);
		float3 wi;
		float pdf;
		float3 BRDF;
		if ((glass == 0)) {
			BRDF = brdf.Sample(wo, &wi, &pdf, uniformSample);
		//	rtPrintf("%f \n", brdf.getR().x);
		}
		else {
			BRDF = brdfGlass.Sample(wo, &wi, &pdf, uniformSample);
			//float cost = brdfGlass.F.SnellCosine(AbsCosTheta(wo));
		//	float r= brdfGlass.F.Reflectivity(CosTheta(wo), cost);
			//rtPrintf("%f %f %f %f %f %f\n", wi.x, wi.y, wi.z, wo.x,wo.y,wo.z);
		}
		float3 wiW = onb.LocalToWorld(wi);
		Ray radianceRay;
		if (glass==0)
			 radianceRay = make_Ray(hit.position + 0.00*hit.normal, wiW, RayType::RADIANCE, 0, RT_DEFAULT_MAX);
		else {
			//rtPrintf("%f %f %f %f %f %f \n", hit.position.x,hit.position.y,hit.position.z, wiW.x,wiW.y,wiW.z);
			float3 hitP = hit.position;
			hitP.y += 0.01*wiW.y;

			radianceRay = make_Ray(hitP, wiW, RayType::RADIANCE, 0, RT_DEFAULT_MAX);
		}
		ReinhartPayload newReinhartPayload;
		newReinhartPayload.depth = reinhartPayload.depth + 1;
		newReinhartPayload.rng = reinhartPayload.rng;
		rtTrace(root, radianceRay, newReinhartPayload);
		//float nDotWi = fabsf(wi.z);
		float nDotWi = fabsf(dot(wiW,hit.normal));
		value = BRDF * nDotWi * newReinhartPayload.value / pdf;
		reinhartPayload.patchID = newReinhartPayload.patchID;

		//rtPrintf("%f %f %f\n", BRDF.x, BRDF.y, BRDF.z);
	}
	reinhartPayload.value = value;
}

RT_PROGRAM void anyHit() {

	shadowPayload.attenuation = 0.0f;
	rtTerminateRay();

}

RT_PROGRAM void miss() {

	//reinhartPayload.value = make_double3(1.0,1.0,1.0);
	reinhartPayload.value = make_float3(1.0f);
	reinhartPayload.patchID = reinhart(ray.direction, 1);

	if (reinhartPayload.patchID == 0) {
		//float t = -ray.origin.y / ray.direction.y;
		//rtPrintf(" %f %f %f %f %f %f \n", (ray.origin + t* ray.direction).x, (ray.origin + t* ray.direction).y, (ray.origin + t* ray.direction).z, ray.direction.x, ray.direction.y, ray.direction.z);

		//rtPrintf("%d %f %f %f %f %f %f \n", reinhartPayload.depth, ray.origin.x, ray.origin.y, ray.origin.z, ray.direction.x, ray.direction.y, ray.direction.z);

		//rtPrintf(" %f %f %f    %f %f %f\n", ray.origin.x, ray.origin.y, ray.origin.z, (ray.origin + 10 * ray.direction).x, (ray.origin + 10 * ray.direction).y, (ray.origin + 10 * ray.direction).z);

	//	rtPrintf("(%f %f %f)  (%f %f %f) %d %f \n   ", ray.origin.x, ray.origin.y, ray.origin.z, ray.direction.x, ray.direction.y, ray.direction.z, reinhartPayload.patchID, reinhartPayload.value.x);
		//rtPrintf("%d\n",  reinhartPayload.patchID);

		//rtPrintf("%f %f %f   \n   ", ray.origin.x, ray.origin.y, ray.origin.z);*/ 
	}
}

