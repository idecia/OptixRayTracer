#include "hip/hip_runtime.h"
#include "lights/AllLights.h"
#include "core/Ray.h"
#include "cameras/Pinhole.h"
#include "samplers\Random2D.h"
#include "samplers\Hemisphere.h"
#include "core/Payload.h"
#include "brdfs/Lambertian.h"
#include "core/optix_global.h"
#include <optix_device.h>

#define MAX_DEPTH 0

rtDeclareVariable(uint2, pixelIdx, rtLaunchIndex, );
rtDeclareVariable(Pinhole, camera, , );
rtDeclareVariable(HitRecord, hit, attribute hit, );
rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(Lambertian, brdf, , );
rtDeclareVariable(ShadowPayload, shadowPayload, rtPayload, );
rtDeclareVariable(RadiancePayload, radiancePayload, rtPayload, );
rtDeclareVariable(rtObject, root, , );
rtBuffer<Light*> lights;

RT_PROGRAM void closestHit() {

   float3 color = make_float3(0.0f);
   BRDFQueryRecord  dummy;

   //ambient light
   //float3 color = brdf.Eval(dummy) * 0.1; 

   //direct light
   unsigned int numLights = lights.size();
   int nSamples = numLights;
   Random2D sampler(&radiancePayload.rng, nSamples);
   for (unsigned int i = 0; i < numLights; i++) {
      Light* light = lights[i];
      float2 uniformSample;
      //while (sampler.Next2D(&uniformSample)) {
	  sampler.Next2D(&uniformSample);
         float3 wo, L; 
         float pdf, tMax;
		 CALL_LIGHT_VIRTUAL_FUNCTION(L, = , light, Sample, hit.position, uniformSample, wo, pdf, tMax);
       //  L = light->Sample(hit.position, uniformSample,  wo, pdf, tMax);
         /*float3 lightPoint = light.Sample(uniformSample);
         float3 L = lightPoint - hit.position;
         float LDist = length(L);
         L = normalize(L);*/
         ShadowPayload shadowPayload;
         shadowPayload.attenuation = 1.0f;
         Ray shadowRay = make_Ray(hit.position, wo, RayType::SHADOW, 0.1, tMax);
         rtTrace(root, shadowRay, shadowPayload);
         if (shadowPayload.attenuation > 0.0f) {
            /*float nDotl  = fmaxf(dot(hit.normal, L), 0.0f);
            float lpDotl = fmaxf(dot(light.NormalAt(lightPoint), -L), 0.0f);
            float3 BRDF = brdf.Eval(dummy);
            float pdf = light.Pdf(lightPoint);
            color += BRDF * nDotl * lpDotl * light.GetColor() / (LDist * LDist * pdf);*/
            float3 BRDF = brdf.Eval(dummy);
            float nDotl = fmaxf(dot(hit.normal, wo), 0.0f);
            color += BRDF * nDotl * L  / pdf;
         }
     // }
      
   }
   color /= (float)nSamples;
   
   
   //indirect light
   if (radiancePayload.depth < MAX_DEPTH) {

      Hemisphere hsSampler(&radiancePayload.rng, 1);
      Onb onb(hit.normal);
      float3 direction;
      hsSampler.Next3D(&direction);
      //cosine_sample_hemisphere(radiancePayload.rng.RandomFloat(), radiancePayload.rng.RandomFloat(), direction);
      onb.inverse_transform(direction);
      float nDotl = fmaxf(dot(hit.normal, direction), 0.0f);
      float3 BRDF = brdf.Eval(dummy);
      float pdf = brdf.Pdf(dummy);
      Ray radianceRay = make_Ray(hit.position, direction, RayType::RADIANCE, 0.5, RT_DEFAULT_MAX);
      RadiancePayload pl;
      pl.depth = radiancePayload.depth + 1;
      pl.rng = radiancePayload.rng;
      rtTrace(root, radianceRay, pl);
       color += BRDF * nDotl * pl.color / pdf;
      //color += BRDF * pl.color;

   }
   
   radiancePayload.color = color;

   

}

RT_PROGRAM void anyHit() {

   shadowPayload.attenuation = 0.0f;
   rtTerminateRay();

}

RT_PROGRAM void miss() {

   float3 BLACK = make_float3(0.0f);
   radiancePayload.color = BLACK;

}

