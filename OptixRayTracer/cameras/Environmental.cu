#include "hip/hip_runtime.h"

#include <optix_world.h>

#include "core/optix_global.h"
#include "core/Ray.h"
#include "core/Payload.h"
#include "samplers/Util.h"
#include "core/RNG.h"
#include "core/ONB.h"
#include "core/math.h"
#include "skyes/Beckers288.h"
#include "samplers/Random2D.h"
#include <optix_device.h>

rtBuffer<RNG> rngs;
rtBuffer<float3, 2> coeff ;
rtDeclareVariable(uint, pixelIdx, rtLaunchIndex, );
rtDeclareVariable(float3, sensorNormal, , );
rtDeclareVariable(float3, sensorPos, , );
rtDeclareVariable(rtObject, root, , );

RT_PROGRAM void sensor(void) {

	int nSamples = 1000;
	ReinhartPayload pl;
	pl.rng = rngs[pixelIdx];

	Random2D sampler(&pl.rng, nSamples);
	float2 unifSample; 

	while (sampler.Next2D(&unifSample)) {
		pl.value = make_float3(0.0f);
		pl.depth = 0;

		ONB onb(sensorNormal);
		float3 dir  = UniformHemisphereSample(unifSample.x, unifSample.y);
		float3 dirW = onb.LocalToWorld(dir);
		Ray ray = make_Ray(sensorPos, dirW, RayType::RADIANCE, 0, RT_DEFAULT_MAX);
		rtTrace(root, ray, pl);

		if (fmaxf(pl.value) > 0.0) {
			uint2 index;
			index.x = beckers(dir);
			index.y = pl.patchID;
			float3 value = pl.value;
			atomicAdd(&coeff[index].x, (float)value.x);
			atomicAdd(&coeff[index].y, (float)value.y);
			atomicAdd(&coeff[index].z, (float)value.z); // falta multiplicar por 2pi/N
		}


	}
	rngs[pixelIdx] = pl.rng;

}


RT_PROGRAM void exception() {

	rtPrintExceptionDetails();

}
