#include "hip/hip_runtime.h"
#include "skyes/Beckers288.h"
#include "core/Ray.h"
#include "samplers/Random2D.h"
#include "core/Payload.h"
#include "bxdfs/Lambertian.h"
#include "lights/EnvironmentLight.h"
#include "core/optix_global.h"
#include "core/ONB.h"
#include "core/math.h"
#include <optix_device.h>

#define MAX_DEPTH 4


rtDeclareVariable(uint, pixelIdx, rtLaunchIndex, );
rtDeclareVariable(HitRecord, hit, attribute hit, );
rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(Lambertian, brdf, , );
rtDeclareVariable(float3, point, , );
rtDeclareVariable(float3, sensorNormal, , );
rtDeclareVariable(BeckersPayload, beckersPayload, rtPayload, );
rtDeclareVariable(ShadowPayload, shadowPayload, rtPayload, );
rtDeclareVariable(rtObject, buildingWindows, , );
rtDeclareVariable(EnvironmentLight, light, , );
rtBuffer<float3> coeff;
rtDeclareVariable(float3, Le, , );

/*
RT_PROGRAM void closestHit() {

double3 value = make_double3(0.0, 0.0, 0.0);
ONB onb(hit.normal);
float3 woW = -ray.direction;
float3 wo = onb.WorldToLocal(woW);

if (BeckersPayload.depth < MAX_DEPTH) {
Random2D sampler(&BeckersPayload.rng, 1);
float2 uniformSample;
sampler.Next2D(&uniformSample);
float3 wi;
float pdf;
//float3 BRDF = brdf.Sample(wo, &wi, &pdf, uniformSample);
float3 wiW = onb.LocalToWorld(wi);
double3 BRDF = ToDouble(brdf.Sample(wo, &wi, &pdf, uniformSample));
double3 wiWd = ToDouble(wiW);
Ray radianceRay = make_Ray(hit.position + 0.0001*hit.normal, wiW, RayType::RADIANCE, 0, RT_DEFAULT_MAX);
BeckersPayload newBeckersPayload;
newBeckersPayload.depth = BeckersPayload.depth + 1;
newBeckersPayload.rng = BeckersPayload.rng;
rtTrace(root, radianceRay, newBeckersPayload);
//float nDotWi = fabsf(wi.z);
//float nDotWi = fabsf(dot(wiW,hit.normal));
double3 normal = ToDouble(hit.normal);
double nDotWi = fabs(dot(wiWd, normal));
//value = BRDF * nDotWi * newBeckersPayload.value / pdf;
value = BRDF * nDotWi * newBeckersPayload.value / (double)pdf;
BeckersPayload.patchID = newBeckersPayload.patchID;

//rtPrintf("%f %f %f\n", BRDF.x, BRDF.y, BRDF.z);
}
BeckersPayload.value = value;
}
*/


/*RT_PROGRAM void closestHit2() {

	ONB onb(hit.normal);
	ONB onbSensor(sensorNormal);
	float3 woW = -ray.direction;
	float3 wo = onb.WorldToLocal(woW);

	if ((beckersPayload.depth == 0) && (Le.x > 0)) {
		float3 value = beckersPayload.value;
		float3 dirID = beckers(onbSensor.WorldToLocal(-woW));
		atomicAdd(&coeff[dirID].x, (float)value.x);
		atomicAdd(&coeff[dirID].y, (float)value.y);
		atomicAdd(&coeff[dirID].z, (float)value.z);
		return;
	}


	//direct light
	unsigned int nLights = 1;
	int nSamples = 1;
	Random2D sampler(&beckersPayload.rng, nSamples);
	for (unsigned int i = 0; i < nLights; i++) {
		float2 uniformSample;
		sampler.Next2D(&uniformSample);
		float3 wiW, L;
		float pdf, tMax;
		L = light.Sample(hit.position, uniformSample, wiW, pdf, tMax);
		ShadowPayload shadowPayload;
		shadowPayload.attenuation = 1.0f;
		Ray shadowRay = make_Ray(hit.position, wiW, RayType::BECKERS_SHADOW, 0.01, tMax);
		rtTrace(buildingWindows, shadowRay, shadowPayload);
		if (shadowPayload.attenuation > 0.0f) {
			float3 wi = onb.WorldToLocal(wiW);
			float3 BRDF = brdf.Eval(wo, wi);
			float nDotWi = fmaxf(wi.z, 0.0f);
			float3 value = beckersPayload.value * BRDF * nDotWi * L / pdf;
			flaot3 dirID = beckers(onbSensor.WorldToLocal(wiW));
			atomicAdd(&coeff[dirID].x, (float)value.x);
			atomicAdd(&coeff[dirID].y, (float)value.y);
			atomicAdd(&coeff[dirID].z, (float)value.z);
		}
	}

	if (beckersPayload.depth < MAX_DEPTH) {
		Random2D sampler(&beckersPayload.rng, 1);
		float2 uniformSample;
		sampler.Next2D(&uniformSample);
		float3 wi;
		float pdf;
		float3 BRDF;

		BRDF = brdf.Sample(wo, &wi, &pdf, uniformSample);
		//	rtPrintf("%f \n", brdf.getR().x);

		float3 wiW = onb.LocalToWorld(wi);
		Ray radianceRay;
		radianceRay = make_Ray(hit.position + 0.00*hit.normal, wiW, RayType::BECKERS_RADIANCE, 0, RT_DEFAULT_MAX);

		BeckersPayload newBeckersPayload;
		newBeckersPayload.depth = beckersPayload.depth + 1;
		newBeckersPayload.rng = beckersPayload.rng;
		newBeckersPayload.value = beckersPayload.value * BRDF * nDotWi * pdf;
		rtTrace(buildingWindows, radianceRay, newBeckersPayload);
	}
}
*/

RT_PROGRAM void closestHit() {

	ONB onb(hit.normal);
	ONB onbSensor(sensorNormal);
	float3 woW = -ray.direction;
	float3 wo = onb.WorldToLocal(woW);
	if (Le.x > 0) {
		beckersPayload.value = make_float3(1.0);
		beckersPayload.dirID = beckers(onbSensor.WorldToLocal(-woW));
		return;
	}


	float3 value = make_float3(0.0f);
	if (beckersPayload.depth < MAX_DEPTH) {
		Random2D sampler(&beckersPayload.rng, 1);
		float2 uniformSample;
		sampler.Next2D(&uniformSample);
		float3 wi;
		float pdf;
		float3 BRDF;

		BRDF = brdf.Sample(wo, &wi, &pdf, uniformSample);
		//	rtPrintf("%f \n", brdf.getR().x);

		float3 wiW = onb.LocalToWorld(wi);
		Ray radianceRay;
		radianceRay = make_Ray(hit.position + 0.00*hit.normal, wiW, RayTypeOpt::BECKERS_RADIANCE, 0, RT_DEFAULT_MAX);

		BeckersPayload newBeckersPayload;
		newBeckersPayload.depth = beckersPayload.depth + 1;
		newBeckersPayload.rng = beckersPayload.rng;
		rtTrace(buildingWindows, radianceRay, newBeckersPayload);
		//float nDotWi = fabsf(wi.z);
		float nDotWi = fabsf(dot(wiW, hit.normal));
		value = BRDF * nDotWi * newBeckersPayload.value / pdf;
		beckersPayload.dirID = newBeckersPayload.dirID;

		//rtPrintf("%f %f %f\n", BRDF.x, BRDF.y, BRDF.z);
	}
	beckersPayload.value = value;
}



RT_PROGRAM void anyHit() {

	shadowPayload.attenuation = 0.0f;
	rtTerminateRay();

}

RT_PROGRAM void miss() {

	//BeckersPayload.value = make_double3(1.0,1.0,1.0);

	beckersPayload.value = make_float3(0.0);
	beckersPayload.dirID = 0;
	//if (BeckersPayload.patchID == 0) {
	//float t = -ray.origin.y / ray.direction.y;
	//rtPrintf(" %f %f %f %f %f %f \n", (ray.origin + t* ray.direction).x, (ray.origin + t* ray.direction).y, (ray.origin + t* ray.direction).z, ray.direction.x, ray.direction.y, ray.direction.z);

	//rtPrintf("%d %f %f %f %f %f %f \n", BeckersPayload.depth, ray.origin.x, ray.origin.y, ray.origin.z, ray.direction.x, ray.direction.y, ray.direction.z);

	//rtPrintf(" %f %f %f    %f %f %f\n", ray.origin.x, ray.origin.y, ray.origin.z, (ray.origin + 10 * ray.direction).x, (ray.origin + 10 * ray.direction).y, (ray.origin + 10 * ray.direction).z);

	//	rtPrintf("(%f %f %f)  (%f %f %f) %d %f \n   ", ray.origin.x, ray.origin.y, ray.origin.z, ray.direction.x, ray.direction.y, ray.direction.z, BeckersPayload.patchID, BeckersPayload.value.x);
	//rtPrintf("%d\n",  BeckersPayload.patchID);

	//rtPrintf("%f %f %f   \n   ", ray.origin.x, ray.origin.y, ray.origin.z);*/ 
	//}
}

