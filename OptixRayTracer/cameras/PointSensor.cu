#include "hip/hip_runtime.h"

#include <optix_world.h>

#include "core/optix_global.h"
#include "core/Ray.h"
#include "core/Payload.h"
#include "samplers/Util.h"
#include "core/RNG.h"
#include "core/ONB.h"
#include "core/math.h"
#include "samplers/Random2D.h"
#include <optix_device.h>

rtBuffer<RNG> rngs;
//rtBuffer<float2> u;
rtBuffer<float3> coeff;
//rtBuffer<double3> coeff;
rtBuffer<unsigned int> ns;
rtDeclareVariable(int, N, ,);
rtDeclareVariable(uint, pixelIdx, rtLaunchIndex, );
rtDeclareVariable(float3, sensorNormal, , );
rtDeclareVariable(float3, sensorPos, , );
rtDeclareVariable(rtObject, root, , );
rtDeclareVariable(unsigned int, NskyPatches, , );


RT_PROGRAM void sensor(void) {

	int nSamples = 500;
	ReinhartPayload pl;
	pl.rng = rngs[pixelIdx];

	Random2D sampler(&pl.rng, nSamples);
	float2 unifSample; //= u[pixelIdx];

	while (sampler.Next2D(&unifSample)) {

		//pl.value = make_float3(0.0f);
		pl.value = make_double3(0.0,0.0,0.0);
		pl.depth = 0;


		ONB onb(sensorNormal);
		float3 dir   = UniformHemisphereSample(unifSample.x, unifSample.y);
		float3 dirW  = onb.LocalToWorld(dir);
		Ray ray = make_Ray(sensorPos, dirW, RayType::RADIANCE, 0, RT_DEFAULT_MAX);
		//rtPrintf("%f %f %f\n", dirW.x, dirW.y, dirW.z);

		rtTrace(root, ray, pl);
		
		if (fmax(pl.value) > 0.0) {
			double3 value = pl.value  ;
			//rtPrintf("%d %f %f\n", pl.patchID, (float)value.x, (float)value.x);
			atomicAdd(&coeff[pl.patchID].x, (float)value.x);
			atomicAdd(&coeff[pl.patchID].y, (float)value.y);
			atomicAdd(&coeff[pl.patchID].z, (float)value.z);
			//ns[pl.patchID]++;
		}
		
		
	}
	/*for (int i = 0; i < NskyPatches; i++) {
		coeff[i].x /= ns[i];
		coeff[i].y /= ns[i];
		coeff[i].z /= ns[i];
		coeff[i] = coeff[i] * M_PI;
		//rtPrintf("%d %f\n", ns[i], coeff[i].x);
	}*/
	rngs[pixelIdx] = pl.rng;

}


RT_PROGRAM void exception() {

	rtPrintExceptionDetails();

}
