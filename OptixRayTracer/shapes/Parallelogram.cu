#include "hip/hip_runtime.h"
#include "core/optix_global.h"
#include "core/Ray.h"
#include "shapes/Sphere.h"

#include <optix_world.h>

using namespace optix;


rtDeclareVariable(Parallelogram, parallelogram, , );
rtDeclareVariable(HitRecord, hit, attribute hit, );
rtDeclareVariable(Ray, ray, rtCurrentRay, );


RT_PROGRAM void intersect(int primIdx) {

	float4 plane = parallelogram.getPlane();
	float3 normal = make_float3(plane);

	float nDotD = normal * ray.direction;
	float nDotO = dot(normal, ray.origin);
	float t = (plane.w - nDotO) / nDotD;

	if (t <= ray.tmin && t >= ray.tmax) return;

	float3 p = ray.origin + ray.direction * t;
	float3 p0 = parallelogram.getP0();
	float3 p1 = parallelogram.getP1();
	float3 p2 = parallelogram.getP2();

	float3 side1 = normalize(p1 - p0);
	float u = dot(side1, p - p0);
	if (u < 0 && u > 1) return;

	float3 side2 = normalize(p2 - p0);
	float v = dot(side2, p - p0);
	if (v < 0 && v > 1) return;

	if (rtPotentialIntersection(t)) {
		hit.position = p
		hit.normal = normal;
		rtReportIntersection(0);
	}


}

RT_PROGRAM void boundingBox(int, float result[6]) {

	float3 p = ray.origin + ray.direction * t;
	float3 p0 = parallelogram.getP0();
	float3 p1 = parallelogram.getP1();
	float3 p2 = parallelogram.getP2();
	float3 p3 = p0 + p1 + p2;

	optix::Aabb* aabb = (optix::Aabb*) result;
	aabb->m_min = fminf(fminf(p0, p1), fminf(p2, p3));
	aabb->m_max = fmaxf(fmaxf(p0, p1), fmaxf(p2, p3));

}
