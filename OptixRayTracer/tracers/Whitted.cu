#include "hip/hip_runtime.h"
#include "lights/PointLight.h"
#include "core/Ray.h"
#include "cameras/Pinhole.h"
#include "core/Payload.h"
#include "brdfs/Lambertian.h"
#include "core/optix_global.h"
#include <optix_device.h>


rtDeclareVariable(uint2, pixelIdx, rtLaunchIndex, );
rtDeclareVariable(Pinhole, camera, , );
rtDeclareVariable(HitRecord, hit, attribute hit, );
rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(Lambertian, brdf, , );
rtDeclareVariable(ShadowPayload, shadowPayload, rtPayload, );
rtDeclareVariable(RadiancePayload, radiancePayload, rtPayload, );
rtDeclareVariable(rtObject, root, , );
rtBuffer<PointLight> lights;

RT_PROGRAM void closestHit() {

	//float3 color = make_float3(0.0f);
	BRDFQueryRecord  dummy;
	float3 color = brdf.Eval(dummy) * 0.1; //simulate ambient light}
	unsigned int numLights = lights.size();
	for (unsigned int i = 0; i < numLights; i++) {

		PointLight light = lights[i];
		float3 L = light.GetPosition() - hit.position;
		float LDist = length(L);
		L = normalize(L);
		ShadowPayload shadowPayload;
		shadowPayload.attenuation = 1.0f;
		Ray shadowRay = make_Ray(hit.position, L, RayType::SHADOW, 0.1, LDist);
		rtTrace(root, shadowRay, shadowPayload);
		if (shadowPayload.attenuation > 0.0f) {
			float nDotl = dot(hit.normal, L);
			float3 BRDF = brdf.Eval(dummy);
			color += BRDF * nDotl * light.GetColor();
		}
	}
	//color.x = 1; color.y = color.z = 0;
	radiancePayload.color = color;
	//rtPrintf("(%f, %f, %f)\n", hit.position.x, hit.position.y, hit.position.z);
	
}

RT_PROGRAM void anyHit() {

	shadowPayload.attenuation = 0.0f;
	rtTerminateRay();

}

RT_PROGRAM void miss() {

	float3 BLACK = make_float3(0.0f);
	radiancePayload.color = BLACK;

}

