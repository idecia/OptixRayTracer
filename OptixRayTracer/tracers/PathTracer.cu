#include "hip/hip_runtime.h"
#include "lights/AreaLight.h"
#include "core/Ray.h"
#include "cameras/Pinhole.h"
#include "samplers\Random2D.h"
#include "samplers\Hemisphere.h"
#include "core/Payload.h"
#include "brdfs/Lambertian.h"
#include "core/optix_global.h"
#include <optix_device.h>

#define MAX_DEPTH 5

rtDeclareVariable(uint2, pixelIdx, rtLaunchIndex, );
rtDeclareVariable(Pinhole, camera, , );
rtDeclareVariable(HitRecord, hit, attribute hit, );
rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(Lambertian, brdf, , );
rtDeclareVariable(ShadowPayload, shadowPayload, rtPayload, );
rtDeclareVariable(RadiancePayload, radiancePayload, rtPayload, );
rtDeclareVariable(rtObject, root, , );
rtBuffer<AreaLight> lights;

RT_PROGRAM void closestHit() {

	//float3 color = make_float3(0.0f);
	BRDFQueryRecord  dummy;

	//ambient light
	float3 color = brdf.Eval(dummy) * 0.1; 

	//direct light
	unsigned int numLights = lights.size();
	int nSamples = 1;
	Random2D sampler(&radiancePayload.rng, nSamples);
	for (unsigned int i = 0; i < numLights; i++) {
		AreaLight light = lights[i];
		float2 uniformSample;
		while (sampler.Next2D(&uniformSample)) {
			float3 lightPoint = light.Sample(uniformSample);
			float3 L = lightPoint - hit.position;
			float LDist = length(L);
			L = normalize(L);
			ShadowPayload shadowPayload;
			shadowPayload.attenuation = 1.0f;
			Ray shadowRay = make_Ray(hit.position, L, RayType::SHADOW, 0.5, LDist);
			rtTrace(root, shadowRay, shadowPayload);
			if (shadowPayload.attenuation > 0.0f) {
				float nDotl  = fmaxf(dot(hit.normal, L), 0.0f);
				float lpDotl = fmaxf(dot(light.NormalAt(lightPoint), -L), 0.0f);
				float3 BRDF = brdf.Eval(dummy);
				float pdf = light.Pdf(lightPoint);
				color += BRDF * nDotl * lpDotl * light.GetColor() / (LDist * LDist * pdf);
			}
		}
		
	}
	color /= (float)nSamples;
	

	//indirect light
	if (radiancePayload.depth < MAX_DEPTH) {

		Hemisphere hsSampler(&radiancePayload.rng, 1);
		Onb onb(hit.normal);
		float3 direction;
		hsSampler.Next3D(&direction);
		onb.inverse_transform(direction);
		float nDotl = fmaxf(dot(hit.normal, direction), 0.0f);
		float3 BRDF = brdf.Eval(dummy);
		float pdf = brdf.Pdf(dummy);
		Ray radianceRay = make_Ray(hit.position, direction, RayType::RADIANCE, 0.5, RT_DEFAULT_MAX);
		RadiancePayload pl;
		pl.depth = radiancePayload.depth + 1;
		pl.rng = radiancePayload.rng;
		rtTrace(root, radianceRay, pl);
		color += BRDF * nDotl * pl.color / pdf;

	}

	radiancePayload.color = color;

	

}

RT_PROGRAM void anyHit() {

	shadowPayload.attenuation = 0.0f;
	rtTerminateRay();

}

RT_PROGRAM void miss() {

	float3 BLACK = make_float3(0.0f);
	radiancePayload.color = BLACK;

}

