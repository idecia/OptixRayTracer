#include "hip/hip_runtime.h"
#include "core/optix_global.h"
#include "core/Ray.h"
#include "shapes/Triangle.h"
#include <optix_world.h>

using namespace optix;


rtDeclareVariable(Triangle, triangle, , );
rtDeclareVariable(HitRecord, hit, attribute hit, );
rtDeclareVariable(Ray, ray, rtCurrentRay, );


RT_PROGRAM void intersect(int primIdx) {

	float3 p0 = triangle.GetP0();
	float3 p1 = triangle.GetP1();
	float3 p2 = triangle.GetP2();

	float3 E1 = p1 - p0;
	float3 E2 = p2 - p0;
	operator-(p1, p2);

	float3 S1 = cross(ray.direction, E2);
	float invDivisor = 1.0f / dot(S1, E1);

	float3 T = ray.origin - p0;
	float u = dot(T, S1) * invDivisor;
	
	float3 S2 = cross(T, E1);
	float v = dot(ray.direction, S2) * invDivisor;

	if (u < 0.0f || v < 0.0f || u + v > 1.0f) return;

	float t = dot(E2, S2) * invDivisor;

	if (rtPotentialIntersection(t)) {
		float3 p = ray.origin + ray.direction * t;
		hit.normal = triangle.NormalAt(p);
		hit.position = p + 0.5*hit.normal;
		rtReportIntersection(0);

	}



}

RT_PROGRAM void boundingBox(int, float result[6]) {

	float3 p0 = triangle.GetP0();
	float3 p1 = triangle.GetP1();
	float3 p2 = triangle.GetP2();

	optix::Aabb* aabb = (optix::Aabb*) result;
	aabb->m_min = fminf(fminf(p0, p1), p2);
	aabb->m_max = fmaxf(fmaxf(p0, p1), p2);

}



