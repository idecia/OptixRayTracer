#include "hip/hip_runtime.h"
#include "skyes/Reinhart145.h"
#include "core/Ray.h"
#include "samplers/Random2D.h"
#include "core/Payload.h"
#include "bxdfs/Lambertian.h"
#include "bxdfs/ThinGlass.h"
#include "core/optix_global.h"
#include "core/ONB.h"
#include "core/math.h"
#include <optix_device.h>


#define MAX_DEPTH 15



rtDeclareVariable(uint, pixelIdx, rtLaunchIndex, );
rtDeclareVariable(HitRecord, hit, attribute hit, );
rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(Lambertian, brdf, , );
rtDeclareVariable(uint, glass, , );
rtDeclareVariable(ThinGlass, brdfGlass, , );
rtDeclareVariable(float3, point, , );
rtDeclareVariable(ReinhartPayload, reinhartPayload, rtPayload, );
rtDeclareVariable(ShadowPayload, shadowPayload, rtPayload, );
rtDeclareVariable(rtObject, root, , );
/*
RT_PROGRAM void closestHit() {
	
	double3 value = make_double3(0.0, 0.0, 0.0);
	ONB onb(hit.normal);
	float3 woW = -ray.direction;
	float3 wo = onb.WorldToLocal(woW);

	if (reinhartPayload.depth < MAX_DEPTH) {
		Random2D sampler(&reinhartPayload.rng, 1);
		float2 uniformSample;
		sampler.Next2D(&uniformSample);
		float3 wi;
		float pdf;
		//float3 BRDF = brdf.Sample(wo, &wi, &pdf, uniformSample);
		float3 wiW = onb.LocalToWorld(wi);
		double3 BRDF = ToDouble(brdf.Sample(wo, &wi, &pdf, uniformSample));
		double3 wiWd = ToDouble(wiW);
		Ray radianceRay = make_Ray(hit.position + 0.0001*hit.normal, wiW, RayType::RADIANCE, 0, RT_DEFAULT_MAX);
		ReinhartPayload newReinhartPayload;
		newReinhartPayload.depth = reinhartPayload.depth + 1;
		newReinhartPayload.rng = reinhartPayload.rng;
		rtTrace(root, radianceRay, newReinhartPayload);
		//float nDotWi = fabsf(wi.z);
		//float nDotWi = fabsf(dot(wiW,hit.normal));
		double3 normal = ToDouble(hit.normal);
		double nDotWi = fabs(dot(wiWd, normal));
		//value = BRDF * nDotWi * newReinhartPayload.value / pdf;
		value = BRDF * nDotWi * newReinhartPayload.value / (double)pdf;
		reinhartPayload.patchID = newReinhartPayload.patchID;
		
			//rtPrintf("%f %f %f\n", BRDF.x, BRDF.y, BRDF.z);
	}
	reinhartPayload.value = value;
}
*/

RT_PROGRAM void closestHit() {
	float3 value = make_float3(0.0f);
	ONB onb(hit.normal);
	float3 woW = -ray.direction;
	float3 wo = onb.WorldToLocal(woW);

	if (reinhartPayload.depth < MAX_DEPTH) {
		Random2D sampler(&reinhartPayload.rng, 1);
		float2 uniformSample;
		sampler.Next2D(&uniformSample);
		float3 wi;
		float pdf;
		float3 BRDF;
		
		BRDF = brdf.Sample(wo, &wi, &pdf, uniformSample);
		//	rtPrintf("%f \n", brdf.getR().x);
		
		float3 wiW = onb.LocalToWorld(wi);
		Ray radianceRay;
		radianceRay = make_Ray(hit.position + 0.00*hit.normal, wiW, RayTypeOpt::REINHART_RADIANCE, 0.000, RT_DEFAULT_MAX);
		
		ReinhartPayload newReinhartPayload;
		newReinhartPayload.depth = reinhartPayload.depth + 1;
		newReinhartPayload.rng = reinhartPayload.rng;
		rtTrace(root, radianceRay, newReinhartPayload);
		//float nDotWi = fabsf(wi.z);
		float nDotWi = fabsf(dot(wiW,hit.normal));
		value = BRDF * nDotWi * newReinhartPayload.value / pdf;
		reinhartPayload.patchID = newReinhartPayload.patchID;

		//rtPrintf("%f %f %f\n", BRDF.x, BRDF.y, BRDF.z);
	}
	reinhartPayload.value = value;
}

RT_PROGRAM void anyHit() {

	shadowPayload.attenuation = 0.0f;
	rtTerminateRay();

}

RT_PROGRAM void miss() {

	//reinhartPayload.value = make_double3(1.0,1.0,1.0);
	
		reinhartPayload.value = make_float3(1.0f);
	reinhartPayload.patchID = reinhart(ray.direction, 2);
	//float t = -ray.origin.y / ray.direction.y;
//rtPrintf(" %f %f %f %f %f %f %d\n", (ray.origin + t* ray.direction).x, (ray.origin + t* ray.direction).y, (ray.origin + t* ray.direction).z, ray.direction.x, ray.direction.y, ray.direction.z, reinhartPayload.depth);
	//if (reinhartPayload.patchID == 0) {
		//float t = -ray.origin.y / ray.direction.y;
		//rtPrintf(" %f %f %f %f %f %f \n", (ray.origin + t* ray.direction).x, (ray.origin + t* ray.direction).y, (ray.origin + t* ray.direction).z, ray.direction.x, ray.direction.y, ray.direction.z);

		//rtPrintf("%d %f %f %f %f %f %f \n", reinhartPayload.depth, ray.origin.x, ray.origin.y, ray.origin.z, ray.direction.x, ray.direction.y, ray.direction.z);

		//rtPrintf(" %f %f %f    %f %f %f\n", ray.origin.x, ray.origin.y, ray.origin.z, (ray.origin + 10 * ray.direction).x, (ray.origin + 10 * ray.direction).y, (ray.origin + 10 * ray.direction).z);

	//	rtPrintf("(%f %f %f)  (%f %f %f) %d %f \n   ", ray.origin.x, ray.origin.y, ray.origin.z, ray.direction.x, ray.direction.y, ray.direction.z, reinhartPayload.patchID, reinhartPayload.value.x);
		//rtPrintf("%d\n",  reinhartPayload.patchID);

		//rtPrintf("%f %f %f   \n   ", ray.origin.x, ray.origin.y, ray.origin.z);*/ 
	//}
}

